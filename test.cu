#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define MAX_ITERATIONS 1
#define TOLERANCE 1e-5

__global__ void normalizeMatrix(float *matrix, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        float sum = 0.0f;
        for (int i = 0; i < size; i++)
        {
            sum += matrix[i * size + idx];
        }
        for (int i = 0; i < size; i++)
        {
            matrix[i * size + idx] /= sum;
        }
    }
}

__global__ void expandMatrix(float *matrix, float *tempMatrix, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        for (int i = 0; i < size; i++)
        {
            for (int j = 0; j < size; j++)
            {
                matrix[idx * size + j] += matrix[idx * size + i] * matrix[i * size + j];
                tempMatrix[idx * size + j] = matrix[idx * size + j];
            }
        }
    }
}

__global__ void inflateMatrix(float *matrix, float *tempMatrix, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        float column_sum = 0.0f;
        for (int i = 0; i < size; i++)
        {
            column_sum += matrix[i * size + idx];
        }
        for (int i = 0; i < size; i++)
        {
            matrix[idx * size + i] = matrix[idx * size + i] * matrix[idx * size + i];
            matrix[idx * size + i] /= column_sum;
            tempMatrix[idx * size + i] = matrix[idx * size + i];
        }
    }
}

__global__ void calculateDifference(float *matrixA, float *matrixB, int size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
    {
        float sumSqDiff = 0.0f;
        for (int i = 0; i < size * size; i++)
        {
            float error = matrixA[i] - matrixB[i];
            sumSqDiff += error * error;
        }
        *diff = sqrtf(sumSqDiff / (size * size));
    }
}

void markovClustering(float *matrix, int size)
{
    float *d_matrix;
    float *d_tempMatrix;
    hipMalloc((void **)&d_matrix, size * size * sizeof(float));
    hipMalloc((void **)&d_tempMatrix, size * size * sizeof(float));
    hipMemcpy(d_matrix, matrix, size * size * sizeof(float), hipMemcpyHostToDevice);

    dim3 block_size(256);
    dim3 grid_size((size + block_size.x - 1) / block_size.x);

    // Normalization
    normalizeMatrix<<<grid_size, block_size>>>(d_matrix, size);
    hipDeviceSynchronize();

    // Initialize variables for convergence check
    float difference = FLT_MAX;
    float *d_difference;
    hipMalloc((void **)&d_difference, sizeof(float));

    // Expansion-Inflation iterations
    while (difference > TOLERANCE)
    {
        expandMatrix<<<grid_size, block_size>>>(d_matrix, d_tempMatrix, size);
        hipDeviceSynchronize();

        inflateMatrix<<<grid_size, block_size>>>(d_matrix, d_tempMatrix, size);
        hipDeviceSynchronize();

        // Calculate difference between matrices
        calculateDifference<<<grid_size, block_size>>>(d_matrix, d_tempMatrix, size, d_difference);
        hipMemcpy(&difference, d_difference, sizeof(float), hipMemcpyDeviceToHost);

        // Swap matrices
        float *temp = d_matrix;
        d_matrix = d_tempMatrix;
        d_tempMatrix = temp;
    }

    hipMemcpy(matrix, d_matrix, size * size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_matrix);
    hipFree(d_tempMatrix);
    hipFree(d_difference);
}

int main()
{
    int size = 4;
    float matrix[] = {
        1.0f, 2.0f, 3.0f, 4.0f,
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f};

    markovClustering(matrix, size);

    printf("Resulting matrix:\n");
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%.4f ", matrix[i * size + j]);
        }
        printf("\n");
    }

    return 0;
}
