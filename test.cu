#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define TOLERANCE 1e-3

__global__ void normalizeMatrix(float *matrix, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        float colSum = 0.0f;
        for (int i = 0; i < size; i++)
        {
            colSum += matrix[i * size + idx];
        }
        if (colSum != 0.0f)
        {
            for (int i = 0; i < size; i++)
            {
                matrix[i * size + idx] /= colSum;
            }
        }
    }
}

__global__ void expandMatrix(float *matrix, float *tempMatrix, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        for (int i = 0; i < size; i++)
        {
            float sum = 0.0f;
            for (int j = 0; j < size; j++)
            {
                sum += matrix[j * size + idx] * matrix[j * size + i];
            }
            tempMatrix[idx * size + i] = sum;
        }
    }
}

__global__ void inflateMatrix(float *matrix, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        for (int j = 0; j < size; j++)
        {
            matrix[idx * size + j] = fminf(matrix[idx * size + j] * matrix[idx * size + j], 1.0e30f);
        }
    }
}

__global__ void calculateDifference(float *matrixA, float *matrixB, int size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)
    {
        float sumSqDiff = 0.0f;
        for (int i = 0; i < size * size; i++)
        {
            float error = matrixA[i] - matrixB[i];
            sumSqDiff += error * error;
        }
        *diff = sqrtf(sumSqDiff / (size * size));
    }
}

void markovClustering(float *matrix, int size)
{
    float *d_matrix;
    float *d_tempMatrix;
    hipMalloc((void **)&d_matrix, size * size * sizeof(float));
    hipMalloc((void **)&d_tempMatrix, size * size * sizeof(float));
    hipMemcpy(d_matrix, matrix, size * size * sizeof(float), hipMemcpyHostToDevice);

    dim3 block_size(256);
    dim3 grid_size((size + block_size.x - 1) / block_size.x);

    // Normalization
    normalizeMatrix<<<grid_size, block_size>>>(d_matrix, size);
    hipDeviceSynchronize();

    // Initialize variables for convergence check
    float difference = FLT_MAX;
    float *d_difference;
    hipMalloc((void **)&d_difference, sizeof(float));

    // Expansion-Inflation iterations
    while (difference > TOLERANCE)
    {
        expandMatrix<<<grid_size, block_size>>>(d_matrix, d_tempMatrix, size);
        hipDeviceSynchronize();

        inflateMatrix<<<grid_size, block_size>>>(d_tempMatrix, size);
        hipDeviceSynchronize();

        normalizeMatrix<<<grid_size, block_size>>>(d_tempMatrix, size);
        hipDeviceSynchronize();

        // Calculate difference between matrices
        calculateDifference<<<grid_size, block_size>>>(d_matrix, d_tempMatrix, size, d_difference);
        hipMemcpy(&difference, d_difference, sizeof(float), hipMemcpyDeviceToHost);

        // Swap matrices
        float *temp = d_matrix;
        d_matrix = d_tempMatrix;
        d_tempMatrix = temp;
    }

    hipMemcpy(matrix, d_matrix, size * size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_matrix);
    hipFree(d_tempMatrix);
    hipFree(d_difference);
}

int main()
{
    int size = 10;
    float matrix[] = {
        0.0f, 1.0f, 0.5f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        1.0f, 0.0f, 0.2f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.5f, 0.2f, 0.0f, 0.8f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.8f, 0.0f, 0.3f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.3f, 0.0f, 0.6f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.6f, 0.0f, 0.4f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.4f, 0.0f, 0.9f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.9f, 0.0f, 0.7f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.7f, 0.0f, 0.1f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.1f, 0.0f};

    markovClustering(matrix, size);

    printf("Resulting matrix:\n");
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%.4f ", matrix[i * size + j]);
        }
        printf("\n");
    }

    return 0;
}
